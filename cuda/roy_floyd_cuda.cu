#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""



__global__ void RoyFloyd(int* matrix, int k, int N) {

	int i = blockIdx.x;
	int j = threadIdx.x;

	if((matrix[i*N + k] + matrix[k*N + j]) < matrix[i*N + j])
		matrix[i*N + j] = matrix[i*N + k] + matrix[k*N + j];	
}


int main(int argc, char **argv)
{	
	int thread_per_block = 512, i, j;
	int N = 100;
	int* cuda_matrix;
	int size = sizeof(int)* N * N;	
	int *matrix = (int*)malloc(sizeof(int)*N*N);
	int *result = (int*)malloc(sizeof(int)*N*N);
	int num_block = ceil((float)(N*N / (thread_per_block)));
	srand(time(NULL));

	for (int i = 0; i < N*N; i++)
		matrix[i] = rand() % 50;
	for (int i = 0; i < N; i++)
		matrix[i*N + i] = 0;
	
	memcpy(result, matrix, sizeof(int)*N*N);

	hipMalloc((void**)&cuda_matrix, size);
	hipMemcpy(cuda_matrix, matrix, size, hipMemcpyHostToDevice);
	
	for (int k = 0; k < N; ++k) {
		RoyFloyd << <num_block, (thread_per_block) >> >(cuda_matrix, k, N);
	}

	hipMemcpy(matrix, cuda_matrix, size, hipMemcpyDeviceToHost);
	hipFree(cuda_matrix);

	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
			printf("dist[%d][%d] = %d \n", i, j, result[i*N + j]);
	}

}